#include "hip/hip_runtime.h"
#include "../bm_config.hpp"

__global__ void raw1f_copy_kernel(float* p_src, float* p_dst, int_t size) {
    for (int_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += blockDim.x * gridDim.x) {
        p_dst[i] = p_src[i];
    }
}

void bm_cuda_raw1f_for_copy(benchmark::State& state) {
    int ts_size = state.range(0);
    cuda::tensor<float, 1> cts_src(ts_size);
    cuda::tensor<float, 1> cts_dst(ts_size);

    cuda::for_index_execution_policy policy;
    policy.total_size(cts_src.size());
    cuda::configure_grid(policy, raw1f_copy_kernel);

    while (state.KeepRunning()) {
        raw1f_copy_kernel<<<policy.grid_dim()[0], policy.block_dim()[0], policy.shared_mem_bytes(),
                            policy.stream()>>>(cts_src.data(), cts_dst.data(), cts_src.size());
        hipDeviceSynchronize();

        benchmark::DoNotOptimize(cts_dst.data());
    }

    state.SetBytesProcessed(state.iterations() * static_cast<size_t>(cts_src.size()) *
                            sizeof(cts_src[0]));
    state.SetItemsProcessed(state.iterations() * static_cast<size_t>(cts_src.size()));
}

BENCHMARK(bm_cuda_raw1f_for_copy)->Arg(1_G);

template <typename tensor_type>
inline void bm_tensor_for_array_index_copy(benchmark::State& state) {
    int ts_size = state.range(0);
    constexpr int_t rank = tensor_type::rank;
    pointi<rank> shape;
    fill(shape, ts_size);

    tensor_type ts_src(shape);
    tensor_type ts_dst(shape);

    while (state.KeepRunning()) {
        cuda::for_index(shape, [ts_src, ts_dst] MATAZURE_GENERAL(pointi<rank> idx) {
            ts_dst(idx) = ts_src(idx);
        });
        // cuda::copy(ts_src, ts_dst);
        benchmark::DoNotOptimize(ts_dst.data());
    }

    state.SetBytesProcessed(state.iterations() * static_cast<size_t>(ts_src.size()) *
                            sizeof(ts_src[0]));
    state.SetItemsProcessed(state.iterations() * static_cast<size_t>(ts_src.size()));
}

auto bm_cuda_tensor1f_for_array_index_copy = bm_tensor_for_array_index_copy<cuda::tensor<float, 1>>;
auto bm_cuda_tensor2f_for_array_index_copy = bm_tensor_for_array_index_copy<cuda::tensor<float, 2>>;
BENCHMARK(bm_cuda_tensor1f_for_array_index_copy)->Arg(1_G);
// cuda中二维的坐标是会耗时更多， 大概有2%的损耗，说明编译器无法将数组访问形式的代码优化点
BENCHMARK(bm_cuda_tensor2f_for_array_index_copy)->Arg(10_K);
