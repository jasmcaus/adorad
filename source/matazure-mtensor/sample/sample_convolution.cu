#include "hip/hip_runtime.h"
﻿#include <mtensor.hpp>
#include "image_helper.hpp"

using namespace matazure;

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cout << "sample6_convolution input_image" << std::endl;
        return -1;
    }

    tensor<pointb<3>, 2> img_rgb = read_rgb_image(argv[1]);
    tensor<pointf<3>, 2> kernel_mean(pointi<2>{3, 3});
    fill(kernel_mean, pointf<3>{0.111f, 0.111f, 0.111f});

    cuda::tensor<pointb<3>, 2> cimg_rgb(img_rgb.shape());
    mem_copy(img_rgb, cimg_rgb);
    cuda::tensor<pointf<3>, 2> ckernel_mean(kernel_mean.shape());
    mem_copy(kernel_mean, ckernel_mean);

    typedef dim<16, 16> BLOCK_DIM;
    pointi<2> block_dim = BLOCK_DIM::value();
    auto valid_block_dim = block_dim - kernel_mean.shape() + pointi<2>{1, 1};
    auto grid_dim = (img_rgb.shape() + valid_block_dim - pointi<2>{1, 1}) / valid_block_dim;
    auto padding = kernel_mean.shape() / 2;

    cuda::tensor<pointf<3>, 2> cimg_mean(img_rgb.shape());

    cuda::block_for_index<BLOCK_DIM>(grid_dim, [=] __device__(
                                                   cuda::block_index<BLOCK_DIM> block_idx) {
        auto valid_global_idx = valid_block_dim * block_idx.block + block_idx.local - padding;
        __shared__ local_tensor<pointf<3>, BLOCK_DIM> sh_ts_block;

        if (inside_rect(valid_global_idx, pointi<2>{0, 0}, cimg_rgb.shape())) {
            sh_ts_block(block_idx.local) = point_cast<float>(cimg_rgb(valid_global_idx));
        } else {
            sh_ts_block(block_idx.local) = zero<pointf<3>>::value();
        }

        cuda::syncthreads();

        if (inside_rect(block_idx.local, padding,
                        block_idx.block_dim - ckernel_mean.shape() + pointi<2>{1, 1}) &&
            inside_rect(valid_global_idx, zero<pointi<2>>::value(), cimg_rgb.shape())) {
            auto sum = zero<pointf<3>>::value();
            for_index(zero<pointi<2>>::value(), ckernel_mean.shape(), [&](const pointi<2>& idx) {
                sum += sh_ts_block(block_idx.local + idx - padding) * ckernel_mean(idx);
            });
            cimg_mean(valid_global_idx) = sum;
        }
    });

    cuda::tensor<pointb<3>, 2> cimg_mean_byte(cimg_mean.shape());
    cuda::transform(cimg_mean, cimg_mean_byte,
                    [] __device__(pointf<3> pixel) { return point_cast<byte>(pixel); });

    tensor<pointb<3>, 2> img_mean(cimg_mean_byte.shape());
    mem_copy(cimg_mean_byte, img_mean);

    write_rgb_png("conv.png", img_mean);

    return 0;
}
