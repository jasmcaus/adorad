#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector>

auto cpuVectorAddition(std::vector<int> A, std::vector<int> B) {
    auto start = std::chrono::high_resolution_clock::now();
    for(int i = 0;i< A.size();i++) {
        A[i] += B[i];
    }
    auto stop = std::chrono::high_resolution_clock::now(); 
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start); 
    std::cout <<"Speed of CPU vector Addition: " << duration.count() <<" micro seconds"<<std::endl; 
    return duration.count();
}

__global__ void add(int *a, int *b, int*c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

// GPU vector Addition using Pointers
auto gpuVectorAddition(std::vector<int> A, std::vector<int> B) {
     
    size_t n= A.size();

    int* h_A = A.data();
    int* h_B = B.data();

    int *d_a, *d_b, *d_c;
    int* h_C = (int *)malloc(sizeof(int)*n);

    hipMalloc((void**)&d_a, sizeof(int)*n);
    hipMalloc((void**)&d_b, sizeof(int)*n);
    hipMalloc((void**)&d_c, sizeof(int)*n);

    hipMemcpy((void *)d_a, h_A, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_b, h_B, sizeof(int)*n, hipMemcpyHostToDevice);

    // Timing stuff, record how many seconds it takes for this operation
    hipEvent_t launch_begin, launch_end;
    hipEventCreate(&launch_begin);
    hipEventCreate(&launch_end);

    // Warmup
    add<<<n,1>>>(d_a, d_b, d_c);// num blocks, num_threads
    float total_time = 0;
    int num_times = 10;
    // Get average of 100 runs
    for(int i = 0;i<num_times;i++) {
        hipEventRecord(launch_begin,0);
        add<<<n,1>>>(d_a, d_b, d_c);
        hipEventRecord(launch_end,0);
        hipEventSynchronize(launch_end);
        float time = 0;
        hipEventElapsedTime(&time, launch_begin, launch_end);
        total_time += time;
    }

    total_time /= num_times;
    std::cout <<"Speed of GPU vector Addition: " << total_time <<" micro seconds"<<std::endl; 
    // Copy memory back and free stuff
    hipMemcpy(h_C, (void **)d_c, sizeof(int)*n, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return total_time;
}

void vectorAdditionSpeedTest() {
    size_t n = 1000000;
    // std::cout<<n<<std::endl;

    std::vector<int> A(n,1);
    std::vector<int> B(n,-1);
    auto timeCpu = cpuVectorAddition(A,B);
    auto timeGpu = gpuVectorAddition(A,B);

    std::cout<<"Speedup over CPU for addition is: "<< (float)timeCpu/timeGpu <<std::endl;
}

// Observation for CPU vs GPU compute in vector addition, the answer why is as follows:

// 1. CUDA has a start-up overhead. For "small" problems like this one, the startup overhead will outweigh any gains from using the GPU. 

template<typename T>
__global__ void mm(T* a, T* b, T* c, T width) {

    int x = blockIdx.x; // block id
    int y = threadIdx.x; // thread id
    T temp = 0;
    for(int i = 0;i< width;i++) {
        temp += a[x*width + i]*b[i*width+ y];
    }

    c[x*width + y] = temp;
}



// GPU vector Addition using Pointers
auto gpuMatrixMultiplication(std::vector<int> A, std::vector<int> B, int size, bool print) {
    
    size_t n= A.size();

    int* h_A = A.data();
    int* h_B = B.data();

    int *d_a, *d_b, *d_c;
    int* h_C = (int *)malloc(sizeof(int)*n);
    hipMalloc((void**)&d_a, sizeof(int)*n);
    hipMalloc((void**)&d_b, sizeof(int)*n);
    hipMalloc((void**)&d_c, sizeof(int)*n);

    hipMemcpy((void *)d_a, h_A, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy((void *)d_b, h_B, sizeof(int)*n, hipMemcpyHostToDevice);

    // Timing stuff, record how many seconds it takes for this operation
    hipEvent_t launch_begin, launch_end;
    hipEventCreate(&launch_begin);
    hipEventCreate(&launch_end);

    // Warmup
    mm<int><<<size,size>>>(d_a, d_b, d_c,size);// num blocks, num_threads
    float total_time = 0;
    int num_times = 10;
    if(!print){
        // Get average of 100 runs
        for(int i = 0;i<num_times;i++) {
            hipEventRecord(launch_begin,0);
            mm<int><<<size,size>>>(d_a, d_b, d_c, size);
            hipEventRecord(launch_end,0);
            hipEventSynchronize(launch_end);
            float time = 0;
            hipEventElapsedTime(&time, launch_begin, launch_end);
            total_time += time;
        }
        
    }

    total_time /= num_times;

    // Copy memory back and free stuff
    hipMemcpy(h_C, (void **)d_c, sizeof(int)*n, hipMemcpyDeviceToHost);

    if(print) {
        for(int i = 0;i < n;i++) {
            std::cout<<h_C[i]<<" ";
        }
        std::cout<<std::endl;
    } else {
        std::cout <<"Speed of GPU vector Multiplication: " << total_time <<" micro seconds"<<std::endl; 
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return total_time;
}

float mmCpu(std::vector<int> &a, std::vector<int> &b, int n,bool print) {

    std::vector<int> c(n*n,0);
    auto start = std::chrono::high_resolution_clock::now();

    for(int i = 0;i<n;i++) {
        for(int j = 0;j< n;j++) {
            for(int k = 0;k < n;k++) {
                c[i*n + k] += a[i*n + j] * b[j*n + k];
            }
        }
    }

    auto stop = std::chrono::high_resolution_clock::now(); 
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start); 

    if(print) {
        for(auto i: c) {
            std::cout<<i<<" ";
        }
        std::cout<<std::endl;
    } else {
        std::cout <<"Speed of CPU vector Multiplication: " << duration.count() <<" micro seconds"<<std::endl; 
    }

    return duration.count();
}

void matrixMultiplySpeedTest() {
    int size = 100;
    std::vector<int> A(size*size);
    std::vector<int> B(size*size);
    auto gpuSpeed = gpuMatrixMultiplication(A,B,size,false);
    auto cpuSpeed = mmCpu(A,B,size,false);

    std::cout<<"Speed of GPU over CPU is: "<< cpuSpeed/gpuSpeed<<std::endl;
}


void matrixMultiplyCorrectness() {
    int size = 4;
    std::vector<int> A = {3,1,2,4,3,1,2,4,3,1,2,4,3,1,2,4};
    std::vector<int> B = {3,1,2,4,3,1,2,4,3,1,2,4,3,1,2,4};
    gpuMatrixMultiplication(A,B,size,true);
    mmCpu(A,B,size,true);
}

// int main() {
//     // vectorAdditionSpeedTest();
//     matrixMultiplySpeedTest();
//     matrixMultiplyCorrectness();
//     return 1; 
// }

